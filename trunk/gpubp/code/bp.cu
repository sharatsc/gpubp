#include "hip/hip_runtime.h"
#include "common.h"
#include "bp.h"
#include <ctype.h>
#include <time.h>

#include <fstream>
#include <iostream>
#include <sstream>
#include <iomanip>
#include <algorithm>
#include "hip/hip_runtime.h"
#include "cutil.h"

#define NTHREADS 4 


__device__ void dgetidx(const int *psizes,int* uidx,int npar,int m)
{
  int stride[MAX_PARENTS];
  stride[0]=1;
  for(int i=1;i<npar;i++)
	stride[i]=stride[i-1]*psizes[i];
  for(int n=(npar-1);n>=0;n--)
	{
	  uidx[n]=m/stride[n];
	  m-=(uidx[n]*stride[n]);
	}
}

void copy_network(network** ppdst,network* psrc,int mode,int alloc=1)
{
  network h_copy;
  if(mode==TOGPU)
	{
	  memcpy(&h_copy,psrc,sizeof(network));

	  for(int i=0;i<psrc->num;i++)
		{
		  if(alloc)
			{
			  float * ptr;
			  hipMalloc((void**)&(h_copy.var[i].M),psrc->var[i].Msize*sizeof(float));
			}
		    hipMemcpy(h_copy.var[i].M,psrc->var[i].M,sizeof(float)*psrc->var[i].Msize,hipMemcpyHostToDevice);
		}
		if(alloc)
		  hipMalloc((void**)ppdst,sizeof(network));
		hipMemcpy(*ppdst,&h_copy,sizeof(network),hipMemcpyHostToDevice);
	}
  else //FROM GPU
	{
	  /*make sure you free resources*/
	  if(alloc)
		*ppdst=(network*)malloc(sizeof(network));
	  hipMemcpy(*ppdst,psrc,sizeof(network),hipMemcpyDeviceToHost);
	  
	  network* pdst=*ppdst;
	  for(int i=0;i<pdst->num;i++)
		{
		  float* ptr=pdst->var[i].M;
  		  pdst->var[i].M=(float*)malloc(pdst->var[i].Msize*sizeof(float));
		  hipMemcpy(pdst->var[i].M,ptr,pdst->var[i].Msize*sizeof(float),hipMemcpyDeviceToHost);
		}
	}
}

void free_gpu_network_resources(network **ppnet)
{
  network net;
  hipMemcpy(&net,*ppnet,sizeof(network),hipMemcpyDeviceToHost);
  for(int i=0;i<net.num;i++)
	{
	  hipFree(net.var[i].M);
	}
  hipFree(*ppnet);
  *ppnet=0;
}

typedef frame* frameptr;

__global__ void dummy(frameptr pframe)
{
  for(int i=0;i<MAX_NODES;i++)
  {
    frameptr ptr=&pframe[i];
    ptr->cidx[0]=1;
    ptr->psizes[0]=10;
  }
}
__global__ void update_belief(void* netptr,void* fptr,void* srcptr,void* dstptr)
{
  int tidx=threadIdx.x;
  int n=blockIdx.x;
  /*determine indices from children*/
  int p,c,i,x;
  network* pnet  =(network*)netptr;
  message* g_src =(message*)srcptr;
  message* g_dst =(message*)dstptr;
   frame*  g_frame=(frame*)fptr;

  frame* pframe=&g_frame[n];
  int nchild=pnet->var[n].nchild;
  int npar  =pnet->var[n].npar;
  int size  =pnet->var[n].size;

  /*determine boundaries of each thread*/
  int csize = (nchild+NTHREADS-1)/NTHREADS;
  int cstart= tidx*csize;
  int cstop = min(cstart+csize,nchild);

  int psize = (npar+NTHREADS-1)/NTHREADS;
  int pstart= tidx*psize;
  int pstop = min(pstart+psize,npar);

  int xsize = (size+NTHREADS-1)/NTHREADS;
  int xstart= tidx*xsize;
  int xstop = min(xstart+xsize,size);
  node* np=&(pnet->var[n]);
  
  for(int c=cstart;c<cstop;c++)
	{
	  node* child=&(pnet->var[np->children[c]]);
	  for(p=0;p<child->npar;p++)
	  {
	  	if(child->parents[p]==n)
	    {
		  pframe->cidx[c]=p;
		  break;
		}
	  }
	}
  __syncthreads();
  /*determine indices from parents*/
  for(p=pstart;p<pstop;p++)
	{
	  node* parent=&(pnet->var[np->parents[p]]);
	  for(c=0;c<parent->nchild;c++)
	    {
		  if(parent->children[c]==n)
			{
			  pframe->pidx[p]=c;
			  break;
			}
		}
	}
  __syncthreads();
  /*size of parents*/
  if(tidx==0)
	{
	  for(pframe->msize=1,p=0;p<np->npar;p++)
		{
		  node* parent=&(pnet->var[np->parents[p]]);
		  pframe->psizes[p]=parent->size;
		  pframe->msize*=pframe->psizes[p];
		}
	}
  __syncthreads();

  /*update lambda*/
  for(x=xstart;x<xstop;x++)
	{
	  pframe->lambda[x]=1;
	  if(np->nchild==0)
		{
		  pframe->lambda[x]=np->lambda[x];
		}
	  else
		{
		  for(c=0;c<np->nchild;c++)
			{
			  float cval=g_src->lambda[np->children[c]][pframe->cidx[c]][x];
			  pframe->lambda[x]*=cval;
			}
		}
	}
   __syncthreads();
  /*update pi*/
  for(x=xstart;x<xstop;x++)
	{
	  pframe->pi[x]=0;
	  if(np->npar==0)
		{
		  pframe->pi[x]=np->M[x];
		}
	  else
		{
		  for(int m=0;m<pframe->msize;m++)
			{
			  float prod=1;
			  dgetidx(pframe->psizes,pframe->uidx,np->npar,m);
			  for(p=0;p<np->npar;p++)
				{
				  prod*=g_src->pi[np->parents[p]][pframe->pidx[p]][pframe->uidx[p]];
				}
			  pframe->pi[x]+=np->M[m+x*pframe->msize]*prod;
			}
		}
	}

  __syncthreads();
  float sum=0;

  /*update belief*/
  if(tidx==0)
	{
	  for(x=0;x<np->size;x++)
		{
		  pframe->bel[x]=pframe->lambda[x]*pframe->pi[x];
		  sum+=pframe->bel[x];
		}
	  for(x=0;x<np->size;x++)
		{
		  pnet->var[n].bel[x]=pframe->bel[x]/sum;
	   }
	}
	__syncthreads();
  /*bottom-up propagation*/
  for(p=0;p<np->npar;p++)
	{
	  int   curpar=np->parents[p];
	  node* parent=&(pnet->var[curpar]);
	  sum=0;
	  for(x=xstart;x<xstop;x++)
		{
		  for(int m=0;m<pframe->msize;m++)
			{
			  float prod=1;
			  dgetidx(pframe->psizes,pframe->uidx,np->npar,m);
			  for(int pp=0;pp<np->npar;pp++)
				{
				  int thispar=np->parents[pp];
				  if(pp==p)continue;
				  prod*=g_src->pi[thispar][pframe->pidx[pp]][pframe->uidx[pp]];
				}
			  pframe->term[n][pframe->uidx[p]][x]+=np->M[x*pframe->msize+m]*prod;
			}
		}
	  __syncthreads();/*wait for everybody to compute sum*/
	  float usum=0;
	  int u=0;
	  if(tidx==0)
		{
		  for(u=0;u<parent->size;u++)
			{
			  sum=0;
			  for(x=0;x<np->size;x++)
				{
				  sum+=pframe->term[n][u][x]*pframe->lambda[x];
				}
			  usum+=sum;
			  g_dst->lambda[n][p][u]=sum;
			}
		  for(int u=0;u<parent->size;u++)
			{
			  g_dst->lambda[n][p][u]/=usum;
			}
		}
	}/*end parent*/
    __syncthreads();
   //top down propagation
	for(c=cstart;c<cstop;c++)
	  {
		int curchild=np->children[c];
		sum=0;
		for(x=0;x<np->size;x++)
		  {
			pframe->py[x]=pframe->bel[x]/(g_src->lambda[curchild][pframe->cidx[c]][x]+1e-5);
			sum+=pframe->py[x];
		  }
		for(x=0;x<np->size;x++)
		  {
			g_dst->pi[n][c][x]=pframe->py[x]/sum;
		  }
	  }
	__syncthreads();
}


void gpu_bnet(network* pnet_h,int niter,int mode=SUMPRODUCT)
{
  network* pnet_g; //gpu_copy
  message* g_src,*g_dst; /*gpu messages*/
  frame *g_frame; /*gpu frames*/
  int f,n,x,c,p,m;

  cout<<"Initializing messages"<<endl;
  /*allocate message structures*/
  CUDA_SAFE_CALL(hipMalloc((void**)&g_src,sizeof(message)));
  CUDA_SAFE_CALL(hipMalloc((void**)&g_dst,sizeof(message)));

  init_messages(&h_src); /*initialize on host*/
  CUDA_SAFE_CALL(hipMemcpy(g_src,&h_src,sizeof(message),hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(g_dst,&h_src,sizeof(message),hipMemcpyHostToDevice));
  cout<<"done"<<endl;

  /*allocate_frame*/
  CUDA_SAFE_CALL(hipMalloc((void**)&g_frame,MAX_NODES*sizeof(frame)));
  CUDA_SAFE_CALL(hipMemset(g_frame,0,MAX_NODES*sizeof(frame)));
  /*copy to gpu*/
  copy_network(&pnet_g,pnet_h,TOGPU,1);
  long start_time=clock();
  for(int iter=0;iter<2;iter++)
	{
	  /*update belief*/
	  dim3 gridSize(3,1,1);
	  dim3 blkSize(10,1,1);
      //dummy<<<gridSize,blkSize>>>(g_frame);
      update_belief<<<gridSize,blkSize>>>(pnet_g,g_frame,g_src,g_dst);
	  CUT_CHECK_ERROR("something bad happened");
	  /*update messages*/
	  CUDA_SAFE_CALL(hipDeviceSynchronize());
	  CUDA_SAFE_CALL(hipMemcpy(g_dst,g_src,sizeof(message),hipMemcpyDeviceToDevice));
  #if 1
      cout<<"--------Frames("<<iter<<")---------"<<endl;
      hipMemcpy(h_frame,g_frame,sizeof(frame)*MAX_NODES,hipMemcpyDeviceToHost);

    for(n=0;n<pnet_h->num;n++)
    { 
	    cout<<"-----------------------------------node("<<n<<")"<<endl;
	    int  sz=pnet_h->var[n].size;

	    cout<<"size:"<<sz<<endl;

        cout<<"cidx:";
	    for(c=0;c<pnet_h->var[n].nchild;c++)
		    cout<<h_frame[n].cidx[c]<<" ";
    	cout<<endl;

	    cout<<"pidx:";
	    for(p=0;p<pnet_h->var[n].npar;p++)
	    	cout<<h_frame[n].pidx[p]<<" ";
	    cout<<endl;

    	cout<<"psize:";
	    for(p=0;p<pnet_h->var[n].npar;p++)
		    cout<<h_frame[n].psizes[p]<<" ";
    	cout<<endl;

	    cout<<"msize:"<<h_frame[n].msize<<endl;

	    cout<<"lambda:";
    	for(x=0;x<sz;x++)
	    {
		    cout<<h_frame[n].lambda[x]<<" ";
	    }
	    cout<<endl;	
  	    cout<<"pi:";
	    for(x=0;x<sz;x++)
	    {
		    cout<<h_frame[n].pi[x]<<" ";
	    }
	    cout<<endl;	
    }	   
    cout<<"---------Beliefs-------"<<endl;
    for(n=0;n<pnet_h->num;n++)
    {
	  cout<<"node"<<n<<":";
	  for(int i=0;i<pnet_h->var[n].size;i++)
		{
		  cout<<pnet_h->var[n].bel[i]<<" ";
		}
	  cout<<endl;
    }
        #endif
  }
  long end_time=clock();
  printf("GPU time elapsed with %d nodes:%.6f\n",pnet_h->num,(float)(end_time-start_time)/CLOCKS_PER_SEC);

  /*copy back to cpu*/
  free_network_resources(pnet_h);
  memset(pnet_h,0,sizeof(network));
  copy_network(&pnet_h,pnet_g,FROMGPU,0);
  cout<<*pnet_h;

 /*clean up*/
  free_gpu_network_resources(&pnet_g);
  hipFree(g_frame);
  hipFree(g_src);
  hipFree(g_dst);
}

